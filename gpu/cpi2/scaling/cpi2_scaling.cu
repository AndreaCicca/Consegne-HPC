
#include <hip/hip_runtime.h>
#include <iostream> 
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
void usage(char * argv[]);

const int threadsPerBlock=256;
int nblocks=128;


__global__ void add( float *res ) {
    __shared__ float shr[threadsPerBlock];


    //// qui ogni thread scrive il proprio indice
    //// da modificare con il calcolo del proprio relativo pezzo di rettangoli
    //// utilizzare blockIdx.x, blockDim.x e gridDim.x per calcolare la propria posizione e le divisioni da gestire

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // step = 1 / dim tot
    float step = 1.0 / (blockDim.x * gridDim.x);
    float x = (tid + 0.5) * step;

    // Calcolo dell'approssimazione di π utilizzando la formula di Gregory-Leibniz
    // integrale che va da 0 a 1 di 1/(1+ x^2) = π/4
    //float height = (4.0f / (1.0f + x*x));
    float height = 4 * sqrt(1-x*x);


    // L'area del rettangolo
    float area = height * step;

    shr[threadIdx.x] = area;

    //shr[threadIdx.x]=threadIdx.x;

    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2 // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
            if (threadIdx.x < i)
        shr[threadIdx.x] += shr[threadIdx.x + i];
        __syncthreads();
        i /= 2;
    }
    if (threadIdx.x==0)
    res[blockIdx.x] = shr[threadIdx.x];
}

// funzione che gestisce flag di input

void options(int argc, char *argv[])
{
  int opt;
  while ((opt = getopt(argc, argv, "n:")) != -1) {
    switch (opt) {
    case 'n':
      nblocks = atoi(optarg);
      break;
    default: /* '?' */
      usage(argv);
      exit(EXIT_FAILURE);
    }
  }
}

int main( int argc, char *argv[] ) { 

    options(argc, argv);

    double  PI = 3.14159265358979323846264338327950288;

    float* res=(float*)malloc(nblocks*sizeof(float));
    float *dev_res;
    HANDLE_ERROR( hipMalloc( (void**)&dev_res, nblocks*sizeof(float) ) );
    //printf("# start\n");

    // calcolo tempo di esecuzione con cudaEvents
    hipEvent_t start;
    hipEvent_t stop;
    float msecTotal;

    hipEventCreate(&start);
    hipEventRecord(start, NULL); 

    add<<<nblocks,threadsPerBlock>>>( dev_res );

    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&msecTotal, start, stop);

    HANDLE_ERROR( hipMemcpy( res, dev_res, nblocks*sizeof(float), hipMemcpyDeviceToHost ) ); 
    
    

    float total=0;
    for (int i=0;i<nblocks;i++){
      //printf("# Block %d: %f\n",i,res[i]);
      total+=res[i];
    }

    printf("# Somma %f\n",total);
    hipFree( dev_res );

    //Numero blocchi, Errore stimato, Tempo di esecuzione divisi da una virgola in stderr
    fprintf(stderr, "# Numero blocchi, Errore stimato, Tempo di esecuzione, somma\n");
    fprintf(stderr, "%d, %lf, %f, %f\n", nblocks, fabs(total-PI), msecTotal, total);
    
    return 0; 
}

void usage(char * argv[])
{
  printf ("\n%s [-n nblocks]",argv[0]);
  printf ("\n");
}
