
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <limits>

static void HandleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

void usage(char * argv[])
{
  printf ("\n%s [-n nblocks]",argv[0]);
  printf ("\n");
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
void usage(char *argv[]);

const int threadsPerBlock = 256;
int nblocks = 128;

__global__ void add(double *res) {
    __shared__ double shr[threadsPerBlock];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    double step = 1.0 / (blockDim.x * gridDim.x);
    double x = (tid + 0.5) * step;

    double height = 4 * sqrt(1 - x * x);
    double area = height * step;

    shr[threadIdx.x] = area;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (threadIdx.x < i)
            shr[threadIdx.x] += shr[threadIdx.x + i];
        __syncthreads();
        i /= 2;
    }
    if (threadIdx.x == 0)
        res[blockIdx.x] = shr[threadIdx.x];
}

void options(int argc, char *argv[]) {
    int opt;
    while ((opt = getopt(argc, argv, "n:")) != -1) {
        switch (opt) {
            case 'n':
                nblocks = atoi(optarg);
                break;
            default: /* '?' */
                usage(argv);
                exit(EXIT_FAILURE);
        }
    }
}

int main(int argc, char *argv[]) {
    options(argc, argv);

    double PI = 3.14159265358979323846264338327950288;

    double *res = (double *)malloc(nblocks * sizeof(double));
    double *dev_res;
    HANDLE_ERROR(hipMalloc((void **)&dev_res, nblocks * sizeof(double)));

    hipEvent_t start;
    hipEvent_t stop;
    float msecTotal;

    hipEventCreate(&start);
    hipEventRecord(start, NULL);

    add<<<nblocks, threadsPerBlock>>>(dev_res);

    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&msecTotal, start, stop);

    HANDLE_ERROR(hipMemcpy(res, dev_res, nblocks * sizeof(double), hipMemcpyDeviceToHost));

    double total = 0;
    for (int i = 0; i < nblocks; i++) {
        total += res[i];
    }

    printf("# Somma %lf\n", total);
    hipFree(dev_res);

    // Precisione massima possibile
    int precision = std::numeric_limits<double>::digits10;
    fprintf(stderr, "# Numero blocchi, Errore stimato, Tempo di esecuzione\n");
    fprintf(stderr, "%d, %.*e, %f \n", nblocks, precision, fabs(total - PI), msecTotal);

    return 0;
}

